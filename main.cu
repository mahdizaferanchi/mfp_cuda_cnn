
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <random>
#include <ctime>
#include <algorithm>
#include <chrono>
#include <string>
#include <array>
#include <functional>

template <class T, size_t S, size_t item_length>
class PinnedData
{
public:
	T* beginning {};
	T* end;
	size_t size = S;
	size_t actual_item_length;
	PinnedData(const std::string& file_name, bool bias=false):
	actual_item_length {item_length + (bias ? 1 : 0)}
	{
		hipHostMalloc((void**) &beginning, sizeof(T) * size * actual_item_length, 4, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		end = beginning;
		std::ifstream file(file_name);
		std::string data_point_string;
		if (actual_item_length == 1)
		{
			while(std::getline(file, data_point_string))
			{
				add_label(data_point_string);
			}
		}else{
			while(std::getline(file, data_point_string))
			{
				add_image(data_point_string);
			}
		}
	}
	void add_image(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		for (int i = 0; i < item_length; ++i)
		{
			end[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
		if (actual_item_length != item_length) 
		{
			end[item_length] = 1.0f;
		}
		end += actual_item_length;
	}
	void add_label(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::getline(line, num, ',');
		end[0] = std::stoi(num);
		end += 1;
	}
	T* operator[](int idx)
	{
		return &beginning[idx * actual_item_length];
	}
};


float get_random_float(float min, float max)
{
    static constexpr double fraction { 1.0 / (RAND_MAX + 1.0) };  // static used for efficiency, so we only calculate this value once
    // evenly distribute the random number across our range
    return min + ((max - min) * (std::rand() * fraction));
}


void fill_with_rand(float* arr, int size, float max=0.1f)
{
	for (int i = 0; i < size; ++i)
	{
		arr[i] = get_random_float(-max, max);
	}
}

class CustomVector
{
public:
	size_t length;
	float* h_copy;
	float* d_copy{};
	CustomVector(size_t p_size, float initial_val=1):
	length{p_size}
	{
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMalloc((void**) &d_copy, float_size * length);
		std::fill_n(h_copy, length, initial_val);
		hipMemcpy(d_copy, h_copy, float_size * length, hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy(h_copy, d_copy, sizeof(float) * length, hipMemcpyDeviceToHost);
		return h_copy;
	}
	friend std::ostream& operator<<(std::ostream& os, CustomVector& vec)
	{
		float* result = vec.read();
		for (int i = 0; i < vec.length; ++i)
		{
			os << result[i] << " ";
		}	
		os << '\n';
		return os;
	}
};

class CustomMatrix
{
public:
	size_t pitch;
	size_t height;
	size_t width;
	float* h_copy;
	float* d_copy{};
	CustomMatrix(size_t p_height, size_t p_width, bool one_initialization=false, float initial_max=0.5):
	height{p_height}, width{p_width}
	{
		size_t length = height * width;
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMallocPitch((void**) &d_copy, &pitch, float_size * width, height);
		if (!one_initialization)
		{
			fill_with_rand(h_copy, length, initial_max);
			for (int i = 0; i < width; ++i)
			{
				h_copy[(height - 1) * width + i] = 0;
			}
		}else{
			std::fill_n(h_copy, length, 1.0f);
		}
		hipMemcpy2D(d_copy, pitch, h_copy, 
			width * float_size, float_size * width, height, 
			hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy2D(h_copy, width * sizeof(float), d_copy, 
			pitch, sizeof(float) * width, height, 
			hipMemcpyDeviceToHost);
		return h_copy;
	}
	void write(float* content)
	{
		size_t float_size = sizeof(float);
		hipMemcpy2D(d_copy, pitch, content, 
			width * float_size, float_size * width, height, 
			hipMemcpyHostToDevice);	
	}
	friend std::ostream& operator<<(std::ostream& os, CustomMatrix& mat)
	{
		float* result = mat.read();
		for (int i = 0; i < mat.height; ++i)
		{
			for (int j = 0; j < mat.width; ++j)
			{
				os << result[i * mat.width + j] << " ";
			}
			os << '\n';
		}	
		return os;
	}
	void print_np()
	{
		float* result = read();
		std::cout << '[';
		for (int i = 0; i < width - 1; ++i)
		{
			std::cout << '[';
			for (int j = 0; j < height; ++j)
			{
				std::cout << result[j * width + i] << ", ";
			}
			std::cout << "], \n";
		}	
		std::cout << "] \n";

		std::cout << '[';
		for(int i = 0; i < height; ++i)
		{
			std::cout << result[i * width + width - 1] << ", ";
		}
		std::cout << "] \n";
	}
	__device__ inline float* at(int row, int col)
	{
		return (float*)((char*)d_copy + row * pitch) + col;
	}
};

class Tensor : public CustomMatrix
{
public:
	size_t height;
	size_t width;
	size_t depth;
	size_t fourth;
	Tensor(size_t p_height, size_t p_width, size_t p_depth=1, size_t p_fourth=1, 
		bool one_initialization=false, float initial_max=0.5):
	CustomMatrix{p_height * p_depth * p_fourth, p_width, one_initialization, initial_max},
	height {p_height}, width {p_width}, depth {p_depth}, fourth {p_fourth}
	{}
	friend std::ostream& operator<<(std::ostream& os, Tensor& mat)
	{
		float* result = mat.read();
		for (int f = 0; f < mat.fourth; ++f)
		{
			os << "block " << f << '\n';
			for (int d = 0; d < mat.depth; ++d)
			{
				os << "depth " << d << '\n';
				for (int i = 0; i < mat.height; ++i)
				{
					for (int j = 0; j < mat.width; ++j)
					{
						os << result[f * mat.depth * mat.height * mat.width + d * mat.height * mat.width + i * mat.width + j] << " ";
					}
					os << '\n';
				}	
			}
		}
		return os;
	}
	__device__ inline float* at(int row, int col, int page=0, int block=0)
	{
		return (float*)((char*)d_copy + (block * height * depth + page * height + row) * pitch) + col;
	}
};

__global__ void matmulvec(float* mat, float* vec, int height, int width, float* out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < height)
	{
		float result = 0.0f;
		for (int j = 0; j < width; ++j)
		{
			result += mat[i * width + j] * vec[j];
		}
		out[i] = result;
	}
}

__global__ void matmulmat(Tensor left, Tensor right, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < left.height && j < right.width)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < left.width; ++loopIdx)
		{
			result += *left.at(i, loopIdx) * (*right.at(loopIdx, j));
		}
		*out.at(i, j) = result;
	}
}

__global__ void matmulmatT(Tensor left, Tensor right, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < left.height && j < right.height)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < left.width; ++loopIdx)
		{
			result += *left.at(i, loopIdx) * (*right.at(j, loopIdx));
		}
		*out.at(i, j) = result;
	}
}

__global__ void transform(Tensor in, Tensor t_mat, Tensor out) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.z * blockDim.z + threadIdx.z;

	__shared__ float intermediate[4][3][1][3];

	if (i < t_mat.height && j < in.width)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < t_mat.width; ++loopIdx)
		{
			result += *t_mat.at(i, loopIdx) * (*in.at(loopIdx, j, k % in.depth, k / in.depth));
		}
		// *out.at(i, j, k % in.depth, k / in.depth) = result;
		intermediate[i][j][k % in.depth][k / in.depth] = result;
	}

	// __syncthreads();

	// if (i < out.height && j < t_mat.height)
	// {
	// 	float result = 0.0f;
	// 	for (int loopIdx = 0; loopIdx < out.width; ++loopIdx)
	// 	{
	// 		result += *out.at(i, loopIdx, k % in.depth, k / in.depth) * (*t_mat.at(j, loopIdx));
	// 	}
	// 	*out.at(i, j, k % in.depth, k / in.depth) = result;
	// }

	if (i < t_mat.height && j < t_mat.height)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < in.width; ++loopIdx)
		{
			result += intermediate[i][loopIdx][k % in.depth][k / in.depth] * (*t_mat.at(j, loopIdx));
		}
		*out.at(i, j, k % in.depth, k / in.depth) = result;
	}

}

__global__ void relu_kernel(Tensor in, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < in.height && j < in.width)
		*out.at(i, j) = (*in.at(i, j) < 0) ? 0 : *in.at(i, j);
}

__global__ void sigmoid_kernel(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i]));
	}
}

__global__ void sigmoid_derivative(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i])) * (1 - 1.0f/(1.0f + expf(-input[i])));
	}
}

__global__ void elementwisemul(Tensor left, Tensor right, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < left.height && j < left.width)
		*out.at(i, j) = *left.at(i, j) * (*right.at(i, j));
}

__global__ void relu_derivative(Tensor in, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
		*out.at(i, j) = (*in.at(i, j) < 0.0f) ? 0.0f : 1.0f;
}

__global__ void softmax_kernel(Tensor in, Tensor out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		float sum = 0;
		for (int loopIdx = 0; loopIdx < in.width; ++loopIdx)
		{
			sum += expf(*in.at(i, loopIdx));
		}
		*out.at(i, j) = expf(*in.at(i, j)) / sum;
	}
}

__global__ void softmax_crossen_error(Tensor in, Tensor out, int* targets)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		if (j == targets[i])
		{
			*out.at(i, j) = *in.at(i, j) - 1;
		}else{
			*out.at(i, j) = *in.at(i, j);
		}
	}
}

__global__ void sigmoid_square_error(Tensor in, Tensor out, int* targets)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		if (i == targets[j])
		{
			*out.at(i, j) = (*in.at(i, j) - 1) * (*in.at(i, j)) * (1 - *in.at(i, j));
		}else{
			*out.at(i, j) = *in.at(i, j) * (*in.at(i, j)) * (1 - *in.at(i, j));
		}
	}
}

__global__ void cross_entropy(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void mean_square_error(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void weight_update_kernel(Tensor errors, Tensor last_activations, Tensor weights, float learning_rate)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < weights.height && j < weights.width)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < errors.height; ++loopIdx)
		{
			result += *last_activations.at(loopIdx, i) * (*errors.at(loopIdx, j));
		}
		*weights.at(i, j) -= learning_rate * result * (1 / (float)errors.height);
	}
}

__global__ void update_correct_labels(Tensor acts, int* labels, int* correct_predictions)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int maxIdx = 0;
	for (int loopIdx = 1; loopIdx < acts.width - 1; ++loopIdx)
	{
		if (*acts.at(i, loopIdx) > *acts.at(i, maxIdx))
			maxIdx = loopIdx;
	}
	if (maxIdx == labels[i])//maxIdx == labels[i]
		atomicAdd(correct_predictions, 1);
}

class Activation
{
public:
	void (*f)(Tensor, Tensor);
	void (*d)(Tensor, Tensor);
	Activation(void (*p_f)(Tensor, Tensor), void (*p_d)(Tensor, Tensor)):
	f{p_f}, d{p_d}
	{} 
};

Activation relu(relu_kernel, relu_derivative);
Activation softmax(softmax_kernel, relu_derivative);
// activation sigmoid(sigmoid_kernel, sigmoid_derivative);

dim3 get_grids(size_t x_dim, size_t y_dim)
{
	return dim3((x_dim > 40) ? x_dim / 20 + 1 : 2, (y_dim > 40) ? y_dim / 20 + 1 : 2);
}

dim3 get_threads(size_t x_dim, size_t y_dim)
{
	return dim3(
		x_dim / ((x_dim > 40) ? x_dim / 20 + 1 : 2) + 1, 
		y_dim / ((y_dim > 40) ? y_dim / 20 + 1 : 2) + 1);
}

class Layer
{
public:
	Tensor activations {1, 1};
	Tensor activations_alt {1, 1};
	Tensor pre_activations {1, 1};
	Tensor errors {1, 1};
	Tensor weights {1, 1};
	Activation act;
	Layer(Activation act_p):
	act {act_p}
	{}
	virtual void forward(Tensor& input, hipStream_t s) = 0;
	virtual void backward(Tensor& nlw, Tensor& nle, hipStream_t s) = 0;
	virtual void set_input_props(const Layer& lla) = 0;
	virtual void initialize_with_batch_size(size_t batch_size, const Layer& ll) = 0;
	virtual size_t get_output_size() const = 0;
	virtual size_t get_output_bias_size() const = 0;
	virtual size_t get_depth() const = 0;
	virtual size_t get_height() const = 0;
	virtual size_t get_width() const = 0;
};


class Regular : public Layer
{
public:
	size_t units;
	size_t input_length;
	bool double_activations;
	Regular(size_t p_units=16, Activation act_p=relu, bool p_double_activations=false, size_t p_input_length=1):
	Layer{act_p}, units{p_units}, input_length{p_input_length}, double_activations{p_double_activations}
	{}
	void forward(Tensor& input, hipStream_t s)
	{
		matmulmat<<<
			get_grids(input.height, units),
			get_threads(input.height, units),
			0, 
			s>>>
			(input, weights, pre_activations);
		act.f<<<
			get_grids(input.height, units),
			get_threads(input.height, units), 
			0, 
			s>>>
			(pre_activations, activations);
	}
	void backward(Tensor& nlw, Tensor& nle, hipStream_t s)
	{
		matmulmatT<<<
			get_grids(nle.height, units), 
			get_threads(nle.height, units), 
			0, 
			s>>>
			(nle, nlw, errors);
		act.d<<<
			get_grids(pre_activations.height, units), 
			get_threads(pre_activations.height, units), 
			0, 
			s>>>
			(pre_activations, pre_activations);
		elementwisemul<<<
			get_grids(errors.height, units), 
			get_threads(errors.height, units), 
			0, 
			s>>>
			(errors, pre_activations, errors);
	}
	void set_input_props(const Layer& ll)
	{
		input_length = ll.get_output_size() + ll.get_output_bias_size();
		weights = Tensor(input_length, units);
	}
	void initialize_with_batch_size(size_t batch_size, const Layer& ll)
	{
		activations = Tensor(batch_size, units + 1, 1, 1, true);
		pre_activations = Tensor(batch_size, units, 1, 1, true);
		errors = Tensor(batch_size, units, 1, 1, true);
		if (double_activations)
			activations_alt = Tensor(batch_size, units + 1, 1, 1, true);
	}
	size_t get_output_size() const
	{
		return units;
	}
	size_t get_output_bias_size() const 
	{
		return 1;
	}
	size_t get_depth() const
	{
		return 1;
	}
	size_t get_height() const
	{
		return activations.height;
	}
	size_t get_width() const
	{
		return activations.width;
	}
};

class Convolutional : public Layer
{
public:
	size_t filter_quantity;
	std::array<size_t, 2> filter_dims;
	std::array<size_t, 2> map_dims;
	Tensor transformed_weights {1, 1};
	bool same_padding {true};
	Convolutional(size_t p_filter_quantity, std::array<size_t, 2> p_filter_dims,
		Activation act_p=relu, bool p_same_padding=true):
	Layer{act_p}, filter_quantity {p_filter_quantity}, same_padding {p_same_padding},
	filter_dims {p_filter_dims}, map_dims {0, 0}
	{}
	Convolutional(size_t p_height, size_t p_width, Activation act_p=relu):
		Layer{act_p}, map_dims {p_height, p_width}, filter_quantity {1}
	{}
	void forward(Tensor& input, hipStream_t s)
	{
		
	}
	void backward(Tensor& nlw, Tensor& nle, hipStream_t s)
	{
		
	}
	void set_input_props(const Layer& ll)
	{
		weights = Tensor(filter_dims[0], filter_dims[1], ll.get_depth(), filter_quantity);
		float filter_transform_matrix_values[12] {1, 0, 0, 0.5, 0.5, 0.5 , 0.5, -0.5, 0.5, 0, 0, 1};
		Tensor G_matrix {4, 3};
		G_matrix.write(filter_transform_matrix_values);
		size_t tile_dim = weights.height + 2 - 1;
		transformed_weights = Tensor(tile_dim, tile_dim, ll.get_depth(), filter_quantity);
		transform<<<
			1, 
			dim3(transformed_weights.height, transformed_weights.width, transformed_weights.depth * transformed_weights.fourth)
			>>>(weights, G_matrix, transformed_weights);
		// std::cout << weights << '\n';
		// std::cout << transformed_weights << '\n';

	}
	void initialize_with_batch_size(size_t batch_size, const Layer& ll)
	{
		const size_t final_height = map_dims[0] ? map_dims[0] : ll.get_height();
		const size_t final_width = map_dims[1] ? map_dims[1] : ll.get_width();
		activations = Tensor(
			final_height, final_width, filter_quantity, batch_size);
		if (map_dims[0])
		{
			activations_alt = Tensor(
				final_height, final_width, filter_quantity, batch_size);
		} else {
			pre_activations = Tensor(
				final_height, final_width, filter_quantity, batch_size);
			errors = Tensor(
				final_height, final_width, filter_quantity, batch_size);
		}
	}
	size_t get_output_size() const
	{
		return activations.height * activations.width * activations.depth;
	}
	size_t get_output_bias_size() const 
	{
		return 0;
	}
	size_t get_depth() const
	{
		// return activations.depth;
		return filter_quantity;
	}
	size_t get_height() const
	{
		return activations.height;
	}
	size_t get_width() const
	{
		return activations.width;
	}
};

typedef void (*out_err_fptr)(Tensor, Tensor, int*);

out_err_fptr get_out_err_func(
	void (*out_loss)(float*, float*, size_t),
	void (*out_act)(Tensor, Tensor))
{
	if (out_loss == cross_entropy)
	{
		if (out_act == softmax_kernel)
		{
			return softmax_crossen_error;
		}else{
			return nullptr;
		}
	}else if (out_loss == mean_square_error){
		if (false)//out_act == sigmoid_kernel
		{
			return sigmoid_square_error;
		}else{
			return nullptr;
		}
	}else{
		return nullptr;
	}
}

class Model
{
public:
	std::vector<std::reference_wrapper<Layer>> layers {};
	int* d_correct_labels {};
	int* d_correct_labels_alt {};
	void (*loss_func)(float*, float*, size_t);
	void (*out_err_func)(Tensor, Tensor, int*);
	bool final {false};
	float learning_rate;
	int* d_correct_predictions {};
	hipStream_t data_transfer_s;
	hipStream_t kernel_exec_s;

	Model(void (*p_loss_func)(float*, float*, size_t), float p_learning_rate):
	loss_func{p_loss_func}, learning_rate{p_learning_rate}
	{
		hipMalloc((void **) &d_correct_predictions, sizeof(int));
		hipStreamCreate(&data_transfer_s);
		hipStreamCreate(&kernel_exec_s);
	}
	void reset_correct_predictions()
	{
		int zero {0};
		hipMemcpy(d_correct_predictions, &zero, sizeof(int), hipMemcpyHostToDevice);
	}
	int read_correct_predictions()
	{
		int ans {};
		hipMemcpy(&ans, d_correct_predictions, sizeof(int), hipMemcpyDeviceToHost);
		return ans;
	}
	bool finalize(size_t batch_size)
	{
		if (get_out_err_func(loss_func, layers.back().get().act.f))
		{
			out_err_func = get_out_err_func(loss_func, layers.back().get().act.f);
			for (std::vector<std::reference_wrapper<Layer>>::iterator l = layers.begin(); l != layers.end(); ++l)
			{
				l->get().initialize_with_batch_size(batch_size, l == layers.begin() ? l->get() : (l - 1)->get());
			}
			hipMalloc((void**) &d_correct_labels, sizeof(int) * batch_size);
			hipMalloc((void**) &d_correct_labels_alt, sizeof(int) * batch_size);
			final = true;
			return true;
		}
		return false;
	}
	void add(Layer& l)
	{
		if(!layers.empty())
		{
			l.set_input_props(layers.back().get());
			layers.push_back(l);
		}else{
			layers.push_back(l);
		}
	}
	void move_batch(float* input_data, int* targets, size_t batch_size, bool use_alt)
	{
		hipMemcpy2DAsync(
			(use_alt ? layers.front().get().activations_alt.d_copy : layers.front().get().activations.d_copy),
			(use_alt ? layers.front().get().activations_alt.pitch : layers.front().get().activations.pitch),
			input_data, 
			sizeof(float) * (layers.front().get().activations.width),
			sizeof(float) * (layers.front().get().activations.width),
			layers.front().get().activations.CustomMatrix::height,
			hipMemcpyHostToDevice,
			data_transfer_s);
		hipMemcpyAsync(
			(use_alt ? d_correct_labels_alt : d_correct_labels), 
			targets, 
			sizeof(int) * batch_size, 
			hipMemcpyHostToDevice,
			data_transfer_s);
	}
	void forward_pass(size_t batch_size, bool use_alt)
	{
		Tensor temp_results = (use_alt ? layers.front().get().activations_alt : layers.front().get().activations);
		for (std::vector<std::reference_wrapper<Layer>>::iterator l = layers.begin() + 1; l != layers.end(); ++l)
		{
			l->get().forward(temp_results, kernel_exec_s);
			temp_results = l->get().activations;
		}
		update_correct_labels<<<1, batch_size, 0, kernel_exec_s>>>(
			layers.back().get().activations, 
			(use_alt ? d_correct_labels_alt : d_correct_labels), 
			d_correct_predictions);
	}
	void backprop(size_t batch_size, bool use_alt)
	{
		out_err_func<<<
		get_grids(batch_size, layers.back().get().get_output_size()), 
		get_threads(batch_size, layers.back().get().get_output_size()), 
		0, 
		kernel_exec_s>>>
		(layers.back().get().activations, 
		layers.back().get().errors, 
		(use_alt ? d_correct_labels_alt : d_correct_labels));
		for (std::vector<std::reference_wrapper<Layer>>::iterator l = layers.end() - 2; l != layers.begin(); --l)
		{
			l->get().backward((l + 1)->get().weights, (l + 1)->get().errors, kernel_exec_s);
		}
	}
	void weight_update(bool use_alt)
	{
		Tensor& input_activations = (use_alt ? layers[0].get().activations_alt : layers[0].get().activations);
		weight_update_kernel<<<
			get_grids(layers[1].get().weights.height, layers[1].get().weights.width),
			get_threads(layers[1].get().weights.height, layers[1].get().weights.width),
			0, 
			kernel_exec_s>>>
			(layers[1].get().errors, input_activations, layers[1].get().weights, learning_rate);
		for (std::vector<std::reference_wrapper<Layer>>::iterator l = layers.begin() + 2; l != layers.end(); ++l)
		{
			weight_update_kernel<<<
				get_grids(l->get().weights.height, l->get().weights.width),
				get_threads(l->get().weights.height, l->get().weights.width),
				0, 
				kernel_exec_s>>>
				(l->get().errors, (l - 1)->get().activations, l->get().weights, learning_rate); 
		}
	}
	void single_train_timed(float* image, int* label, size_t batch_size)
	{
		auto t0 = std::chrono::high_resolution_clock::now();
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		auto t1 = std::chrono::high_resolution_clock::now();
		forward_pass(batch_size, false);
		auto t2 = std::chrono::high_resolution_clock::now();
		backprop(batch_size, false);
		auto t3 = std::chrono::high_resolution_clock::now();
		weight_update(false);
		hipDeviceSynchronize();
		auto t4 = std::chrono::high_resolution_clock::now();
		std::chrono::nanoseconds move_time = t1 - t0;
		std::chrono::nanoseconds forward_time = t2 - t1;
		std::chrono::nanoseconds back_time = t3 - t2;
		std::chrono::nanoseconds update_time = t4 - t1;
		std::cout << move_time.count() << "ns \n";
		std::cout << forward_time.count() << "ns \n";
		std::cout << back_time.count() << "ns \n";
		std::cout << update_time.count() << "ns \n";
	}
	void single_train(float* image, int* label, size_t batch_size)
	{
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		forward_pass(batch_size, false);
		backprop(batch_size, false);
		weight_update(false);
		hipDeviceSynchronize();
	}
	void single_test(float* image, int* label, size_t batch_size)
	{
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		forward_pass(batch_size, false);
		hipDeviceSynchronize();
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void train_sequential(
		PinnedData<T1, S, item_length1> images,
		PinnedData<T2, S, item_length2> labels,
		int epochs,
		size_t batch_size)
	{
		if (finalize(batch_size))
		{
			for (int epoch = 1; epoch <= epochs; ++epoch)
			{
				reset_correct_predictions();
				int num_of_data = images.size;
				auto tik = std::chrono::high_resolution_clock::now();
				for (int loopIdx = 0; loopIdx < num_of_data; loopIdx += batch_size)
				{
					single_train(images[loopIdx], labels[loopIdx], batch_size);
				}
				auto tok = std::chrono::high_resolution_clock::now();
				std::chrono::duration<double, std::milli> ms_double = tok - tik;
				std::cout << "Epoch " << epoch << " : acc = ";
				std::cout << read_correct_predictions()/(float)num_of_data; 
				std::cout << " in " << ms_double.count() << "ms.\n"; 
			}
		}else{
			std::cout << "Could not finalize model. \n";
		}
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void train(
		PinnedData<T1, S, item_length1> images,
		PinnedData<T2, S, item_length2> labels,
		int epochs,
		size_t batch_size)
	{
		if (finalize(batch_size))
		{
			for (int epoch = 1; epoch <= epochs; ++epoch)
			{
				bool use_alt {false};
				reset_correct_predictions();
				int num_of_data = images.size;
				auto tik = std::chrono::high_resolution_clock::now();
				move_batch(images[0], labels[0], batch_size, use_alt);
				hipDeviceSynchronize();
				for (int loopIdx = batch_size; loopIdx < num_of_data; loopIdx += batch_size)
				{
					move_batch(
						images[loopIdx], 
						labels[loopIdx], 
						batch_size, 
						!use_alt);
					forward_pass(batch_size, use_alt);
					backprop(batch_size, use_alt);
					weight_update(use_alt);
					hipDeviceSynchronize();
					use_alt = !use_alt;
				}
				auto tok = std::chrono::high_resolution_clock::now();
				std::chrono::duration<double, std::milli> ms_double = tok - tik;
				std::cout << "Epoch " << epoch << " : acc = ";
				std::cout << read_correct_predictions()/(float)num_of_data; 
				std::cout << " in " << ms_double.count() << "ms.\n"; 
			}
		}else{
			std::cout << "Could not finalize model. \n";
		}
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void test(
		PinnedData<T1, S, item_length1> images,
		PinnedData<T2, S, item_length2> labels,
		size_t batch_size)
	{
		int num_of_data = images.size;
		reset_correct_predictions();
		for (int loopIdx = 0; loopIdx < num_of_data; loopIdx += batch_size)
		{
			single_test(images[loopIdx], labels[loopIdx], batch_size);
		}
		std::cout << "test acc = " << read_correct_predictions()/(float)num_of_data << '\n';
	}
};

__global__ void test(int a, int* b)
{
	*b = a + 1;
}
void testCuda()
{
	int* dp;
	int hp;
	hipMalloc((void**) &dp, sizeof(int));
	test<<<1, 1>>>(1, dp);
	hipMemcpy(&hp, dp, sizeof(int), hipMemcpyDeviceToHost);
	if(hp != 2)
	{
		std::cout << "Cuda Not Working. Factory Reset Runtime. \n";
	}
}

int main()
{
	testCuda();

	std::srand(0);//static_cast<unsigned int>(std::time(nullptr))
	std::rand(); 

	PinnedData<float, 10000, 784> test_images("sample_data/mnist_test.csv");
	PinnedData<int, 10000, 1> test_labels("sample_data/mnist_test.csv");
	PinnedData<float, 20000, 784> train_images("sample_data/mnist_train_small.csv");
	PinnedData<int, 20000, 1> train_labels("sample_data/mnist_train_small.csv");

	// auto layer1 = Regular(784, relu, true);
	auto layer1 = Convolutional(28, 28);
	// auto layer2 = Regular(128);
	auto layer2 = Convolutional(3, {3, 3});
	// auto layer3 = Regular(128);
	auto layer3 = Convolutional(3, {3, 3});
	auto layer4 = Regular(10, softmax);

	Model mnist_model(cross_entropy, 0.05f);
	mnist_model.add(layer1);
	mnist_model.add(layer2);
	mnist_model.add(layer3);
	mnist_model.add(layer4);

	mnist_model.finalize(32);

	std::cout << mnist_model.layers[0].get().weights.depth << '\n';
	std::cout << mnist_model.layers[1].get().weights.depth << '\n';
	std::cout << mnist_model.layers[2].get().weights.depth << '\n';
	std::cout << mnist_model.layers[3].get().weights.depth << '\n';

	std::cout << mnist_model.layers[0].get().activations.depth << '\n';
	std::cout << mnist_model.layers[1].get().activations.depth << '\n';
	std::cout << mnist_model.layers[2].get().activations.depth << '\n';
	std::cout << mnist_model.layers[3].get().activations.depth << '\n';

	// mnist_model.move_batch(train_images[0], train_labels[0], 32, false);
	// cudaDeviceSynchronize();
	// std::cout << cudaGetErrorName(cudaPeekAtLastError()) << '\n';
	// cudaDeviceProp props;
	// cudaGetDeviceProperties(&props, 0);
	// std::cout << props.memPitch << '\n';
	// std::cout << mnist_model.layers.front().get().get_output_size() << '\n';
	// std::cout << mnist_model.layers.front().get().get_output_bias_size() << '\n';
	// std::cout << mnist_model.layers.front().get().activations.pitch << '\n';
	// std::cout << sizeof(float) << '\n';
	// std::cout << mnist_model.layers[0].get().activations << '\n';

	// auto tik = std::chrono::high_resolution_clock::now();
	// mnist_model.train(train_images, train_labels, 7, 32);

	// mnist_model.learning_rate = 0.001f;
	// mnist_model.train(train_images, train_labels, 5, 32);

	// mnist_model.learning_rate = 0.0001f;
	// mnist_model.train(train_images, train_labels, 5, 32);

	// auto tok = std::chrono::high_resolution_clock::now();
	// std::chrono::duration<double, std::milli> ms_double = tok - tik;
	// std::cout << ms_double.count() << "ms \n";

	// mnist_model.test(test_images, test_labels, 32);

	return 0;
}

