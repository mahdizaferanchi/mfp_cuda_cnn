
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <random>
#include <ctime>
#include <algorithm>
#include <chrono>
#include <string>



class mnist_data_point
{
public:
	float image[784];
	int label;
	mnist_data_point(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		label = std::stoi(nums[0]);
		for (int i = 0; i < 784; ++i)
		{
			image[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
	}
};

class mnist_image
{
public:
	float image[785];
	mnist_image(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		for (int i = 0; i < 784; ++i)
		{
			image[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
		image[784] = 1.0f;
	}
};

class mnist_label
{
public:
	int label;
	mnist_label(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::getline(line, num, ',');
		label = std::stoi(num);
	}
};

std::vector<mnist_data_point> mnist_parse(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_data_point> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_data_point p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

std::vector<mnist_image> mnist_parse_image(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_image> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_image p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

std::vector<mnist_label> mnist_parse_label(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_label> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_label p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

float get_random_float(float min, float max)
{
    static constexpr double fraction { 1.0 / (RAND_MAX + 1.0) };  // static used for efficiency, so we only calculate this value once
    // evenly distribute the random number across our range
    return min + ((max - min) * (std::rand() * fraction));
}


void fill_with_rand(float* arr, int size, float max=0.1f)
{
	for (int i = 0; i < size; ++i)
	{
		arr[i] = get_random_float(-max, max);
	}
}

class c_vector
{
public:
	size_t length;
	float* h_copy;
	float* d_copy{};
	c_vector(size_t p_size, float initial_val=1):
	length{p_size}
	{
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMalloc((void**) &d_copy, float_size * length);
		std::fill_n(h_copy, length, initial_val);
		hipMemcpy(d_copy, h_copy, float_size * length, hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy(h_copy, d_copy, sizeof(float) * length, hipMemcpyDeviceToHost);
		return h_copy;
	}
	friend std::ostream& operator<<(std::ostream& os, c_vector& vec)
	{
		float* result = vec.read();
		for (int i = 0; i < vec.length; ++i)
		{
			os << result[i] << " ";
		}	
		os << '\n';
		return os;
	}
};

class c_matrix
{
public:
	size_t pitch;
	size_t height;
	size_t width;
	float* h_copy;
	float* d_copy{};
	c_matrix(size_t p_height, size_t p_width, bool one_initialization=false, float initial_max=0.5):
	height{p_height}, width{p_width}
	{
		size_t length = height * width;
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMallocPitch((void**) &d_copy, &pitch, float_size * width, height);
		if (!one_initialization)
		{
			fill_with_rand(h_copy, length, initial_max);
			for (int i = 0; i < width; ++i)
			{
				h_copy[(height - 1) * width + i] = 0;
			}
		}else{
			std::fill_n(h_copy, length, 1.0f);
		}
		hipMemcpy2D(d_copy, pitch, h_copy, 
			width * float_size, float_size * width, height, 
			hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy2D(h_copy, width * sizeof(float), d_copy, 
			pitch, sizeof(float) * width, height, 
			hipMemcpyDeviceToHost);
		return h_copy;
	}
	friend std::ostream& operator<<(std::ostream& os, c_matrix& mat)
	{
		float* result = mat.read();
		for (int i = 0; i < mat.height; ++i)
		{
			for (int j = 0; j < mat.width; ++j)
			{
				os << result[i * mat.width + j] << " ";
			}
			os << '\n';
		}	
		return os;
	}
	void print_np()
	{
		float* result = read();
		std::cout << '[';
		for (int i = 0; i < width - 1; ++i)
		{
			std::cout << '[';
			for (int j = 0; j < height; ++j)
			{
				std::cout << result[j * width + i] << ", ";
			}
			std::cout << "], \n";
		}	
		std::cout << "] \n";

		std::cout << '[';
		for(int i = 0; i < height; ++i)
		{
			std::cout << result[i * width + width - 1] << ", ";
		}
		std::cout << "] \n";

	}
	__device__ inline float* at(int row, int col)
	{
		return (float*)((char*)d_copy + row * pitch) + col;
	}
};

__global__ void matmulvec(float* mat, float* vec, int height, int width, float* out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < height)
	{
		float result = 0.0f;
		for (int j = 0; j < width; ++j)
		{
			result += mat[i * width + j] * vec[j];
		}
		out[i] = result;
	}
}

__global__ void matmulmat(c_matrix left, c_matrix right, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	float result = 0.0f;
	for (int loopIdx = 0; loopIdx < left.width; ++loopIdx)
	{
		result += *left.at(i, loopIdx) * (*right.at(loopIdx, j));
	}
	*out.at(i, j) = result;
}

__global__ void matTmulvec(float* mat, float* vec, int height, int width, float* out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < width)
	{
		float result = 0.0f;
		for (int j = 0; j < height; ++j)
		{
			result += mat[j * width + i] * vec[j];
		}
		out[i] = result;
	}
}

__global__ void relu_kernel(c_matrix in, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// if (in.at(i, j) < 0)
	// {
	// 	out.at(i, j) = 0;
	// }else{
	// 	out.at(i, j) = in.at(i, j);
	// }
	*out.at(i, j) = (*in.at(i, j) < 0) ? 0 : *in.at(i, j);
}

__global__ void sigmoid_kernel(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i]));
	}
}

__global__ void sigmoid_derivative(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i])) * (1 - 1.0f/(1.0f + expf(-input[i])));
	}
}

__global__ void elementwisemul(float* input1, float* input2, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = input1[i] * input2[i];
	}
}

__global__ void relu_derivative(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (input[i] < 0)
		{
			output[i] = 0;
		}else{
			output[i] = 1;
		}
	}
}

__global__ void softmax_kernel(c_matrix in, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	float sum = 0;
	for (int loopIdx = 0; loopIdx < in.width; ++loopIdx)
	{
		sum += expf(*in.at(i, loopIdx));
	}
	*out.at(i, j) = expf(*in.at(i, j)) / sum;
}

__global__ void softmax_crossen_error(float* input, float* output, size_t size, int target)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (i == target)
		{
			output[i] = (input[i] - 1);
		}else{
			output[i] = input[i];
		}

	}
}

__global__ void sigmoid_square_error(float* input, float* output, size_t size, int target)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		if (i == target)
		{
			output[i] = (input[i] - 1) * input[i] * (1 - input[i]);
		}else{
			output[i] = (input[i]) * input[i] * (1 - input[i]);
		}

	}
}

__global__ void cross_entropy(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void mean_square_error(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void weight_update_kernel(float* errors, float* last_activations, float* weights, float learning_rate)
{
	int i = threadIdx.x * gridDim.x + blockIdx.x;
	weights[i] += (-learning_rate * errors[threadIdx.x] * last_activations[blockIdx.x]);
}

class activation
{
public:
	void (*f)(c_matrix, c_matrix);
	void (*d)(float*, float*, size_t);
	activation(void (*p_f)(c_matrix, c_matrix), void (*p_d)(float*, float*, size_t)):
	f{p_f}, d{p_d}
	{} 
};

activation relu(relu_kernel, relu_derivative);
activation softmax(softmax_kernel, relu_derivative);
// activation sigmoid(sigmoid_kernel, sigmoid_derivative);

class layer
{
public:
	size_t units;
	size_t input_length;
	c_matrix activations {1, 1};
	c_matrix pre_activations {1, 1};
	c_matrix errors {1, 1};
	c_matrix weights {1, 1};
	activation act;
	layer(size_t p_units=16, activation act_p=relu, size_t p_input_length=1):
	units{p_units}, act{act_p}, input_length{p_input_length}
	{}
	void forward(c_matrix input)
	{
		matmulmat<<<dim3(2, 2), dim3(16, 8)>>>(input, weights, pre_activations);
		act.f<<<dim3(2, 2), dim3(16, 8)>>>(pre_activations, activations);
	}
	void backward(c_matrix& nlw, c_matrix& nle)
	{
		matTmulvec<<<2, 8>>>(nlw.d_copy, nle.d_copy, nlw.height, nlw.width, errors.d_copy);
		// act.d<<<2, 8>>>(pre_activations.d_copy, pre_activations.d_copy, pre_activations.length);
		// elementwisemul<<<2, 8>>>(errors.d_copy, pre_activations.d_copy, errors.d_copy, errors.length);
	}
	void set_input_lenght(size_t length)
	{
		input_length = length;
		weights = c_matrix(input_length, units);
	}
	void initialize_with_batch_size(size_t batch_size)
	{
		activations = c_matrix(batch_size, units + 1, true);
		pre_activations = c_matrix(batch_size, units, true);
		errors = c_matrix(batch_size, units, true);
	}
};

typedef void (*out_err_fptr)(float*, float*, size_t, int);

out_err_fptr get_out_err_func(
	void (*out_loss)(float*, float*, size_t),
	void (*out_act)(c_matrix, c_matrix))
{
	if (out_loss == cross_entropy)
	{
		if (out_act == softmax_kernel)
		{
			return softmax_crossen_error;
		}else{
			return nullptr;
		}
	}else if (out_loss == mean_square_error){
		if (false)//out_act == sigmoid_kernel
		{
			return sigmoid_square_error;
		}else{
			return nullptr;
		}
	}else{
		return nullptr;
	}
}

class model
{
public:
	std::vector<layer> layers{};
	float* d_loss{};
	int* d_correct_label{};
	void (*loss_func)(float*, float*, size_t);
	void (*out_err_func)(float*, float*, size_t, int);
	bool final {false};
	float learning_rate;
	float* d_learning_rate;

	model(void (*p_loss_func)(float*, float*, size_t), float p_learning_rate):
	loss_func{p_loss_func}, learning_rate{p_learning_rate}
	{
		hipMalloc((void**) &d_loss, sizeof(float));
		hipMalloc((void**) &d_learning_rate, sizeof(float));
		hipMalloc((void**) &d_correct_label, sizeof(int));
	}
	bool finalize(size_t batch_size)
	{
		if (get_out_err_func(loss_func, layers.back().act.f))
		{
			out_err_func = get_out_err_func(loss_func, layers.back().act.f);
			for (int loopIdx = 0; loopIdx < layers.size(); ++loopIdx)
			{
				layers[loopIdx].initialize_with_batch_size(batch_size);	
			}
			final = true;
			return true;
		}
		return false;
	}
	void add(layer l)
	{
		if(!layers.empty())
		{
			l.set_input_lenght(layers.back().units);
			layers.push_back(l);
		}else{
			layers.push_back(l);
		}
	}
	void forward_pass(float* input_data, size_t batch_size)
	{
		hipMemcpy2D(
			layers.front().activations.d_copy,
			layers.front().activations.pitch,
			input_data, 
			sizeof(float) * (layers.front().units + 1),
			sizeof(float) * (layers.front().units + 1),
			batch_size,
			hipMemcpyHostToDevice);

		// c_matrix temp_results = layers.front().activations;
		// for (std::vector<layer>::iterator l = layers.begin() + 1; l != layers.end(); ++l)
		// {
		// 	l->forward(temp_results);
		// 	temp_results = l->activations;
		// }
	}
	void backprop(int target)
	{
		out_err_func<<<2, 8>>>(layers.back().activations.d_copy, layers.back().errors.d_copy, layers.back().units, target);
		for (std::vector<layer>::iterator l = layers.end() - 2; l != layers.begin(); --l)
		{
			l->backward((l + 1)->weights, (l + 1)->errors);
		}
	}
	// void weight_update()
	// {
	// 	for (std::vector<layer>::iterator l = layers.begin() + 1; l != layers.end(); ++l)
	// 	{
	// 		weight_update_kernel<<<(l - 1)->activations.length, l->errors.length>>>
	// 		(l->errors.d_copy, (l - 1)->activations.d_copy, l->weights.d_copy, learning_rate); 
	// 	}
	// }
	bool single_train(float* image, int* label, size_t batch_size)
	{
		forward_pass(image, batch_size);
		float* result = layers.back().activations.read();
		int prediction = std::max_element(result, result + layers.back().units) - result;
		backprop(*label);
		// weight_update();
		return prediction == *label;
	}
	bool single_test(float* image, int label, size_t batch_size)
	{
		forward_pass(image, batch_size);
		float* result = layers.back().activations.read();
		int prediction = std::max_element(result, result + layers.back().units) - result;
		return prediction == label;
	}
	void train(std::vector<mnist_image>& images,
		std::vector<mnist_label>& labels,
		int epochs,
		size_t batch_size)
	{
		if (finalize(batch_size))
		{
			for (int epoch = 1; epoch <= epochs; ++epoch)
			{
				auto tik = std::chrono::high_resolution_clock::now();
				int num_of_data = images.size();
				float acc = 0;
				for (int loopIdx = 0; loopIdx = num_of_data; loopIdx += batch_size)
				{
					if(single_train(images[loopIdx].image, &labels[loopIdx].label, batch_size))
					{
						acc += 1.0f / num_of_data;
					}
				}
				auto tok = std::chrono::high_resolution_clock::now();
				std::chrono::duration<double, std::milli> ms_double = tok - tik;
				std::cout << "Epoch " << epoch << ": acc = " << acc << " in " << ms_double.count() << "ms.\n"; 
			}
		}else{
			std::cout << "Could not finalize model. \n";
		}
	}
	void test(std::vector<mnist_image>& images, std::vector<mnist_label>& labels)
	{
		int num_of_data = images.size();
		float acc = 0;
		for (int loopIdx = 0; loopIdx < num_of_data; ++loopIdx)
		{
			if(single_test(images[loopIdx].image, labels[loopIdx].label, 1))
			{
				acc += 1.0f / num_of_data;
			}
		}
		std::cout << "test acc = " << acc << '\n';
	}
};

int main()
{
	std::srand(0);//static_cast<unsigned int>(std::time(nullptr))
	std::rand(); 

	auto test_images = mnist_parse_image("sample_data/mnist_test.csv");
	auto test_labels = mnist_parse_label("sample_data/mnist_test.csv");
	auto train_images = mnist_parse_image("sample_data/mnist_train_small.csv");
	auto train_labels = mnist_parse_label("sample_data/mnist_train_small.csv");

	// model mnist_model(mean_square_error, 0.5f);
	// mnist_model.add(layer(784));
	// mnist_model.add(layer(16, sigmoid));
	// mnist_model.add(layer(16, sigmoid));
	// mnist_model.add(layer(10, sigmoid));

	model mnist_model(cross_entropy, 0.01f);
	mnist_model.add(layer(784));
	mnist_model.add(layer(16));
	mnist_model.add(layer(16));
	mnist_model.add(layer(10, softmax));

	mnist_model.finalize(32);

	mnist_model.forward_pass(train_images[0].image, 32);
	mnist_model.layers[1].forward(mnist_model.layers[0].activations);
	std::cout << mnist_model.layers[0].activations << '\n';
	std::cout << mnist_model.layers[1].activations << '\n';

	// auto tik = std::chrono::high_resolution_clock::now();
	// mnist_model.train(train_images, train_labels, 3);
	// auto tok = std::chrono::high_resolution_clock::now();
	// std::chrono::duration<double, std::milli> ms_double = tok - tik;
	// std::cout << ms_double.count() << "ms \n";
	// mnist_model.learning_rate = 0.005f;
	// mnist_model.train(train_data, 5);
	// mnist_model.learning_rate = 0.001f;
	// mnist_model.train(train_data, 5);


	// mnist_model.test(test_images, test_labels);

	return 0;
}

