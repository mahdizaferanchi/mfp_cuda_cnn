
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <sstream>
#include <random>
#include <ctime>
#include <algorithm>
#include <chrono>
#include <string>


template <class T, size_t S, size_t item_length>
class pinned_data
{
public:
	T* beginning {};
	T* end;
	size_t size = S;
	pinned_data(const std::string& file_name)
	{
		hipHostMalloc((void**) &beginning, sizeof(T) * size * item_length, 4, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);
		end = beginning;
		std::ifstream file(file_name);
		std::string data_point_string;
		if (item_length == 1)
		{
			while(std::getline(file, data_point_string))
			{
				add_label(data_point_string);
			}
		}else{
			while(std::getline(file, data_point_string))
			{
				add_image(data_point_string);
			}
		}
	}
	void add_image(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		for (int i = 0; i < 784; ++i)
		{
			end[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
		end[784] = 1.0f;
		end += 785;
	}
	void add_label(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::getline(line, num, ',');
		end[0] = std::stoi(num);
		end += 1;
	}
	T* operator[](int idx)
	{
		return &beginning[idx * item_length];
	}
};

class mnist_data_point
{
public:
	float image[784];
	int label;
	mnist_data_point(std::string& str)
	{
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		label = std::stoi(nums[0]);
		for (int i = 0; i < 784; ++i)
		{
			image[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
	}
};
class mnist_image
{
public:
	float image[785];
	mnist_image(std::string& str)
	{
		hipHostMalloc((void**) &image, 785 * sizeof(float), 4, hipHostMallocDefault);
		std::stringstream line(str);
		std::string num;
		std::vector<std::string> nums;
		while(std::getline(line, num, ','))
		{
			nums.push_back(num);
		}
		for (int i = 0; i < 784; ++i)
		{
			image[i] = static_cast<float> (std::stoi(nums[i + 1])) / 255.0;
		}
		image[784] = 1.0f;
	}
};

class mnist_label
{
public:
	int label[1];
	mnist_label(std::string& str)
	{
		// cudaError_t status = cudaMallocHost((void**) &label, sizeof(int), 4);
		std::stringstream line(str);
		std::string num;
		std::getline(line, num, ',');
		label[0] = std::stoi(num);
	}
};

std::vector<mnist_data_point> mnist_parse(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_data_point> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_data_point p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

std::vector<mnist_image> mnist_parse_image(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_image> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_image p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

std::vector<mnist_label> mnist_parse_label(const std::string& file_name)
{
	std::ifstream file(file_name);
	std::vector<mnist_label> data_vector;
	std::string data_point_string;
	while(std::getline(file, data_point_string))
	{
		mnist_label p(data_point_string);
		data_vector.push_back(p);
	}
	return data_vector;
}

float get_random_float(float min, float max)
{
    static constexpr double fraction { 1.0 / (RAND_MAX + 1.0) };  // static used for efficiency, so we only calculate this value once
    // evenly distribute the random number across our range
    return min + ((max - min) * (std::rand() * fraction));
}


void fill_with_rand(float* arr, int size, float max=0.1f)
{
	for (int i = 0; i < size; ++i)
	{
		arr[i] = get_random_float(-max, max);
	}
}

class c_vector
{
public:
	size_t length;
	float* h_copy;
	float* d_copy{};
	c_vector(size_t p_size, float initial_val=1):
	length{p_size}
	{
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMalloc((void**) &d_copy, float_size * length);
		std::fill_n(h_copy, length, initial_val);
		hipMemcpy(d_copy, h_copy, float_size * length, hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy(h_copy, d_copy, sizeof(float) * length, hipMemcpyDeviceToHost);
		return h_copy;
	}
	friend std::ostream& operator<<(std::ostream& os, c_vector& vec)
	{
		float* result = vec.read();
		for (int i = 0; i < vec.length; ++i)
		{
			os << result[i] << " ";
		}	
		os << '\n';
		return os;
	}
};

class c_matrix
{
public:
	size_t pitch;
	size_t height;
	size_t width;
	float* h_copy;
	float* d_copy{};
	c_matrix(size_t p_height, size_t p_width, bool one_initialization=false, float initial_max=0.5):
	height{p_height}, width{p_width}
	{
		size_t length = height * width;
		size_t float_size = sizeof(float);
		h_copy = new float[length];
		hipMallocPitch((void**) &d_copy, &pitch, float_size * width, height);
		if (!one_initialization)
		{
			fill_with_rand(h_copy, length, initial_max);
			for (int i = 0; i < width; ++i)
			{
				h_copy[(height - 1) * width + i] = 0;
			}
		}else{
			std::fill_n(h_copy, length, 1.0f);
		}
		hipMemcpy2D(d_copy, pitch, h_copy, 
			width * float_size, float_size * width, height, 
			hipMemcpyHostToDevice);
	}
	float* read()
	{
		hipMemcpy2D(h_copy, width * sizeof(float), d_copy, 
			pitch, sizeof(float) * width, height, 
			hipMemcpyDeviceToHost);
		return h_copy;
	}
	friend std::ostream& operator<<(std::ostream& os, c_matrix& mat)
	{
		float* result = mat.read();
		for (int i = 0; i < mat.height; ++i)
		{
			for (int j = 0; j < mat.width; ++j)
			{
				os << result[i * mat.width + j] << " ";
			}
			os << '\n';
		}	
		return os;
	}
	void print_np()
	{
		float* result = read();
		std::cout << '[';
		for (int i = 0; i < width - 1; ++i)
		{
			std::cout << '[';
			for (int j = 0; j < height; ++j)
			{
				std::cout << result[j * width + i] << ", ";
			}
			std::cout << "], \n";
		}	
		std::cout << "] \n";

		std::cout << '[';
		for(int i = 0; i < height; ++i)
		{
			std::cout << result[i * width + width - 1] << ", ";
		}
		std::cout << "] \n";

	}
	__device__ inline float* at(int row, int col)
	{
		return (float*)((char*)d_copy + row * pitch) + col;
	}
};

__global__ void matmulvec(float* mat, float* vec, int height, int width, float* out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < height)
	{
		float result = 0.0f;
		for (int j = 0; j < width; ++j)
		{
			result += mat[i * width + j] * vec[j];
		}
		out[i] = result;
	}
}

__global__ void matmulmat(c_matrix left, c_matrix right, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < left.height && j < right.width)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < left.width; ++loopIdx)
		{
			result += *left.at(i, loopIdx) * (*right.at(loopIdx, j));
		}
		*out.at(i, j) = result;
	}
}

__global__ void matmulmatT(c_matrix left, c_matrix right, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < left.height && j < right.height)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < left.width; ++loopIdx)
		{
			result += *left.at(i, loopIdx) * (*right.at(j, loopIdx));
		}
		*out.at(i, j) = result;
	}
}

__global__ void relu_kernel(c_matrix in, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// if (in.at(i, j) < 0)
	// {
	// 	out.at(i, j) = 0;
	// }else{
	// 	out.at(i, j) = in.at(i, j);
	// }
	if (i < in.height && j < in.width)
		*out.at(i, j) = (*in.at(i, j) < 0) ? 0 : *in.at(i, j);
}

__global__ void sigmoid_kernel(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i]));
	}
}

__global__ void sigmoid_derivative(float* input, float* output, size_t size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size)
	{
		output[i] = 1.0f/(1.0f + expf(-input[i])) * (1 - 1.0f/(1.0f + expf(-input[i])));
	}
}

__global__ void elementwisemul(c_matrix left, c_matrix right, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < left.height && j < left.width)
		*out.at(i, j) = *left.at(i, j) * (*right.at(i, j));
}

__global__ void relu_derivative(c_matrix in, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	// if (i < size)
	// {
	// 	if (input[i] < 0)
	// 	{
	// 		output[i] = 0;
	// 	}else{
	// 		output[i] = 1;
	// 	}
	// }
	if (i < in.height && j < in.width)
		*out.at(i, j) = (*in.at(i, j) < 0.0f) ? 0.0f : 1.0f;
}

__global__ void softmax_kernel(c_matrix in, c_matrix out)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		float sum = 0;
		for (int loopIdx = 0; loopIdx < in.width; ++loopIdx)
		{
			sum += expf(*in.at(i, loopIdx));
		}
		*out.at(i, j) = expf(*in.at(i, j)) / sum;
	}
}

__global__ void softmax_crossen_error(c_matrix in, c_matrix out, int* targets)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		if (j == targets[i])
		{
			*out.at(i, j) = *in.at(i, j) - 1;
		}else{
			*out.at(i, j) = *in.at(i, j);
		}
	}
}

__global__ void sigmoid_square_error(c_matrix in, c_matrix out, int* targets)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < in.height && j < in.width)
	{
		if (i == targets[j])
		{
			*out.at(i, j) = (*in.at(i, j) - 1) * (*in.at(i, j)) * (1 - *in.at(i, j));
		}else{
			*out.at(i, j) = *in.at(i, j) * (*in.at(i, j)) * (1 - *in.at(i, j));
		}
	}
}

__global__ void cross_entropy(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void mean_square_error(float* input, float* output, size_t size)
{
	// int i = blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void weight_update_kernel(c_matrix errors, c_matrix last_activations, c_matrix weights, float learning_rate)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < weights.height && j < weights.width)
	{
		float result = 0.0f;
		for (int loopIdx = 0; loopIdx < errors.height; ++loopIdx)
		{
			result += *last_activations.at(loopIdx, i) * (*errors.at(loopIdx, j));
		}
		*weights.at(i, j) -= learning_rate * result * (1 / (float)errors.height);
	}
}

__global__ void update_correct_labels(c_matrix acts, int* labels, int* correct_predictions)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int maxIdx = 0;
	for (int loopIdx = 1; loopIdx < acts.width - 1; ++loopIdx)
	{
		if (*acts.at(i, loopIdx) > *acts.at(i, maxIdx))
			maxIdx = loopIdx;
	}
	if (maxIdx == labels[i])//maxIdx == labels[i]
		atomicAdd(correct_predictions, 1);
}

class activation
{
public:
	void (*f)(c_matrix, c_matrix);
	void (*d)(c_matrix, c_matrix);
	activation(void (*p_f)(c_matrix, c_matrix), void (*p_d)(c_matrix, c_matrix)):
	f{p_f}, d{p_d}
	{} 
};

activation relu(relu_kernel, relu_derivative);
activation softmax(softmax_kernel, relu_derivative);
// activation sigmoid(sigmoid_kernel, sigmoid_derivative);

class layer
{
public:
	size_t units;
	size_t input_length;
	c_matrix activations {1, 1};
	c_matrix activations_alt {1, 1};
	c_matrix pre_activations {1, 1};
	c_matrix errors {1, 1};
	c_matrix weights {1, 1};
	activation act;
	bool double_activations;
	layer(size_t p_units=16, activation act_p=relu, bool p_double_activations=false, size_t p_input_length=1):
	units{p_units}, act{act_p}, input_length{p_input_length}, double_activations{p_double_activations}
	{}
	void forward(c_matrix& input, hipStream_t s)
	{
		matmulmat<<<dim3(2, 2), dim3(input.height / 2 + 1, units / 2), 0, s>>>(input, weights, pre_activations);
		act.f<<<dim3(2, 2), dim3(input.height / 2 + 1, units / 2), 0, s>>>(pre_activations, activations);
	}
	void backward(c_matrix& nlw, c_matrix& nle, hipStream_t s)
	{
		matmulmatT<<<dim3(2, 2), dim3(nle.height / 2 + 1, units / 2), 0, s>>>(nle, nlw, errors);
		act.d<<<dim3(2, 2), dim3(pre_activations.height / 2 + 1, units / 2), 0, s>>>(pre_activations, pre_activations);
		elementwisemul<<<dim3(2, 2), dim3(errors.height / 2 + 1, units / 2), 0, s>>>(errors, pre_activations, errors);
	}
	void set_input_lenght(size_t length)
	{
		input_length = length;
		weights = c_matrix(input_length, units);
	}
	void initialize_with_batch_size(size_t batch_size)
	{
		activations = c_matrix(batch_size, units + 1, true);
		pre_activations = c_matrix(batch_size, units, true);
		errors = c_matrix(batch_size, units, true);
		if (double_activations)
			activations_alt = c_matrix(batch_size, units + 1, true);
	}
};

typedef void (*out_err_fptr)(c_matrix, c_matrix, int*);

out_err_fptr get_out_err_func(
	void (*out_loss)(float*, float*, size_t),
	void (*out_act)(c_matrix, c_matrix))
{
	if (out_loss == cross_entropy)
	{
		if (out_act == softmax_kernel)
		{
			return softmax_crossen_error;
		}else{
			return nullptr;
		}
	}else if (out_loss == mean_square_error){
		if (false)//out_act == sigmoid_kernel
		{
			return sigmoid_square_error;
		}else{
			return nullptr;
		}
	}else{
		return nullptr;
	}
}

dim3 get_grids(size_t x_dim, size_t y_dim)
{
	return dim3((x_dim > 40) ? x_dim / 20 + 1 : 2, (y_dim > 40) ? y_dim / 20 + 1 : 2);
}

dim3 get_threads(size_t x_dim, size_t y_dim)
{
	return dim3(
		x_dim / ((x_dim > 40) ? x_dim / 20 + 1 : 2) + 1, 
		y_dim / ((y_dim > 40) ? y_dim / 20 + 1 : 2) + 1);
}

class model
{
public:
	std::vector<layer> layers {};
	int* d_correct_labels {};
	int* d_correct_labels_alt {};
	void (*loss_func)(float*, float*, size_t);
	void (*out_err_func)(c_matrix, c_matrix, int*);
	bool final {false};
	float learning_rate;
	int* d_correct_predictions {};
	hipStream_t data_transfer_s;
	hipStream_t kernel_exec_s;

	model(void (*p_loss_func)(float*, float*, size_t), float p_learning_rate):
	loss_func{p_loss_func}, learning_rate{p_learning_rate}
	{
		hipMalloc((void **) &d_correct_predictions, sizeof(int));
		hipStreamCreate(&data_transfer_s);
		hipStreamCreate(&kernel_exec_s);
	}
	void reset_correct_predictions()
	{
		int zero {0};
		hipMemcpy(d_correct_predictions, &zero, sizeof(int), hipMemcpyHostToDevice);
	}
	int read_correct_predictions()
	{
		int ans {};
		hipMemcpy(&ans, d_correct_predictions, sizeof(int), hipMemcpyDeviceToHost);
		return ans;
	}
	bool finalize(size_t batch_size)
	{
		if (get_out_err_func(loss_func, layers.back().act.f))
		{
			out_err_func = get_out_err_func(loss_func, layers.back().act.f);
			for (int loopIdx = 0; loopIdx < layers.size(); ++loopIdx)
			{
				layers[loopIdx].initialize_with_batch_size(batch_size);	
			}
			hipMalloc((void**) &d_correct_labels, sizeof(int) * batch_size);
			hipMalloc((void**) &d_correct_labels_alt, sizeof(int) * batch_size);
			final = true;
			return true;
		}
		return false;
	}
	void add(layer l)
	{
		if(!layers.empty())
		{
			l.set_input_lenght(layers.back().units + 1);
			layers.push_back(l);
		}else{
			layers.push_back(l);
		}
	}
	void move_batch(float* input_data, int* targets, size_t batch_size, bool use_alt)
	{

		// auto t0 = std::chrono::high_resolution_clock::now();
		hipMemcpy2DAsync(
			(use_alt ? layers.front().activations_alt.d_copy : layers.front().activations.d_copy),
			(use_alt ? layers.front().activations_alt.pitch : layers.front().activations.pitch),
			input_data, 
			sizeof(float) * (layers.front().units + 1),
			sizeof(float) * (layers.front().units + 1),
			batch_size,
			hipMemcpyHostToDevice,
			data_transfer_s);
		// auto t1 = std::chrono::high_resolution_clock::now();
		// std::chrono::nanoseconds dt = t1 - t0;
		// std::cout << dt.count() << '\n';
		hipMemcpyAsync(
			(use_alt ? d_correct_labels_alt : d_correct_labels), 
			targets, 
			sizeof(int) * batch_size, 
			hipMemcpyHostToDevice,
			data_transfer_s);
	}
	void forward_pass(size_t batch_size, bool use_alt)
	{
		c_matrix temp_results = (use_alt ? layers.front().activations_alt : layers.front().activations);
		for (std::vector<layer>::iterator l = layers.begin() + 1; l != layers.end(); ++l)
		{
			l->forward(temp_results, kernel_exec_s);
			temp_results = l->activations;
		}
		update_correct_labels<<<1, batch_size, 0, kernel_exec_s>>>(
			layers.back().activations, 
			(use_alt ? d_correct_labels_alt : d_correct_labels), 
			d_correct_predictions);
	}
	void backprop(size_t batch_size, bool use_alt)
	{
		out_err_func<<<dim3(2, 2), dim3(batch_size / 2 + 1, layers.back().units / 2), 0, kernel_exec_s>>>(
			layers.back().activations, 
			layers.back().errors, 
			(use_alt ? d_correct_labels_alt : d_correct_labels));
		for (std::vector<layer>::iterator l = layers.end() - 2; l != layers.begin(); --l)
		{
			l->backward((l + 1)->weights, (l + 1)->errors, kernel_exec_s);
		}
	}
	void weight_update(bool use_alt)
	{
		c_matrix& input_activations = (use_alt ? layers[0].activations_alt : layers[0].activations);
		int dim = (layers[1].weights.height > 50) ? 20 : 2;
		weight_update_kernel<<<
			dim3(dim, 2),
			dim3(layers[1].weights.height/dim + 1, layers[1].weights.width/2 + 1),
			0, 
			kernel_exec_s>>>
		(layers[1].errors, input_activations, layers[1].weights, learning_rate);
		for (std::vector<layer>::iterator l = layers.begin() + 2; l != layers.end(); ++l)
		{
			int var = (l->weights.height > 50) ? 20 : 2;
			weight_update_kernel<<<
				dim3(var, 2),
				dim3(l->weights.height/var + 1, l->weights.width/2 + 1),
				0, 
				kernel_exec_s>>>
			(l->errors, (l - 1)->activations, l->weights, learning_rate); 
		}
	}
	void single_train_timed(float* image, int* label, size_t batch_size)
	{
		auto t0 = std::chrono::high_resolution_clock::now();
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		auto t1 = std::chrono::high_resolution_clock::now();
		forward_pass(batch_size, false);
		// auto t2 = std::chrono::high_resolution_clock::now();
		backprop(batch_size, false);
		// auto t3 = std::chrono::high_resolution_clock::now();
		weight_update(false);
		hipDeviceSynchronize();
		auto t4 = std::chrono::high_resolution_clock::now();
		std::chrono::nanoseconds move_time = t1 - t0;
		// std::chrono::nanoseconds forward_time = t2 - t1;
		// std::chrono::nanoseconds back_time = t3 - t2;
		std::chrono::nanoseconds update_time = t4 - t1;
		std::cout << move_time.count() << "ns \n";
		// std::cout << forward_time.count() << "ns \n";
		// std::cout << back_time.count() << "ns \n";
		std::cout << update_time.count() << "ns \n";
	}
	void single_train(float* image, int* label, size_t batch_size)
	{
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		forward_pass(batch_size, false);
		backprop(batch_size, false);
		weight_update(false);
		hipDeviceSynchronize();
	}
	void single_test(float* image, int* label, size_t batch_size)
	{
		move_batch(image, label, batch_size, false);
		hipDeviceSynchronize();
		forward_pass(batch_size, false);
		hipDeviceSynchronize();
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void train(
		pinned_data<T1, S, item_length1> images,
		pinned_data<T2, S, item_length2> labels,
		int epochs,
		size_t batch_size)
	{
		if (finalize(batch_size))
		{
			for (int epoch = 1; epoch <= epochs; ++epoch)
			{
				reset_correct_predictions();
				int num_of_data = images.size;
				auto tik = std::chrono::high_resolution_clock::now();
				for (int loopIdx = 0; loopIdx < num_of_data; loopIdx += batch_size)
				{
					single_train(images[loopIdx], labels[loopIdx], batch_size);
				}
				auto tok = std::chrono::high_resolution_clock::now();
				std::chrono::duration<double, std::milli> ms_double = tok - tik;
				std::cout << "Epoch " << epoch << " : acc = ";
				std::cout << read_correct_predictions()/(float)num_of_data; 
				std::cout << " in " << ms_double.count() << "ms.\n"; 
			}
		}else{
			std::cout << "Could not finalize model. \n";
		}
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void train_pipelined(
		pinned_data<T1, S, item_length1> images,
		pinned_data<T2, S, item_length2> labels,
		int epochs,
		size_t batch_size)
	{
		if (finalize(batch_size))
		{
			for (int epoch = 1; epoch <= epochs; ++epoch)
			{
				bool use_alt {false};
				reset_correct_predictions();
				int num_of_data = images.size;
				auto tik = std::chrono::high_resolution_clock::now();
				move_batch(images[0], labels[0], batch_size, use_alt);
				hipDeviceSynchronize();
				for (int loopIdx = batch_size; loopIdx < num_of_data; loopIdx += batch_size)
				{
					move_batch(
						images[loopIdx], 
						labels[loopIdx], 
						batch_size, 
						!use_alt);
					// cudaDeviceSynchronize();
					forward_pass(batch_size, use_alt);
					backprop(batch_size, use_alt);
					weight_update(use_alt);
					hipDeviceSynchronize();
					use_alt = !use_alt;
				}
				auto tok = std::chrono::high_resolution_clock::now();
				std::chrono::duration<double, std::milli> ms_double = tok - tik;
				std::cout << "Epoch " << epoch << " : acc = ";
				std::cout << read_correct_predictions()/(float)num_of_data; 
				std::cout << " in " << ms_double.count() << "ms.\n"; 
			}
		}else{
			std::cout << "Could not finalize model. \n";
		}
	}
	template <typename T1, typename T2, size_t S, size_t item_length1, size_t item_length2>
	void test(
		pinned_data<T1, S, item_length1> images,
		pinned_data<T2, S, item_length2> labels,
		size_t batch_size)
	{
		int num_of_data = images.size;
		reset_correct_predictions();
		for (int loopIdx = 0; loopIdx < num_of_data; loopIdx += batch_size)
		{
			single_test(images[loopIdx], labels[loopIdx], batch_size);
		}
		std::cout << "test acc = " << read_correct_predictions()/(float)num_of_data << '\n';
	}
};

int main()
{
	std::srand(0);//static_cast<unsigned int>(std::time(nullptr))
	std::rand(); 

	// auto test_images = mnist_parse_image("sample_data/mnist_test.csv");
	// auto test_labels = mnist_parse_label("sample_data/mnist_test.csv");
	// auto train_images = mnist_parse_image("sample_data/mnist_train_small.csv");
	// auto train_labels = mnist_parse_label("sample_data/mnist_train_small.csv");

	pinned_data<float, 10000, 785> test_images("sample_data/mnist_test.csv");
	pinned_data<int, 10000, 1> test_labels("sample_data/mnist_test.csv");
	pinned_data<float, 20000, 785> train_images("sample_data/mnist_train_small.csv");
	pinned_data<int, 20000, 1> train_labels("sample_data/mnist_train_small.csv");

	// model mnist_model(mean_square_error, 0.5f);
	// mnist_model.add(layer(784));
	// mnist_model.add(layer(16, sigmoid));
	// mnist_model.add(layer(16, sigmoid));
	// mnist_model.add(layer(10, sigmoid));

	model mnist_model(cross_entropy, 0.05f);
	mnist_model.add(layer(784, relu, true));
	mnist_model.add(layer(16));
	mnist_model.add(layer(16));
	mnist_model.add(layer(10, softmax));

	mnist_model.finalize(32);
	// mnist_model.train_pipelined(train_images, train_labels, 10, 32);
	// mnist_model.single_train_timed(train_images[0], train_labels[0], 32);

	auto tik = std::chrono::high_resolution_clock::now();

	mnist_model.train_pipelined(train_images, train_labels, 7, 32);

	// mnist_model.learning_rate = 0.001f;
	// mnist_model.train_pipelined(train_images, train_labels, 5, 32);

	// mnist_model.learning_rate = 0.0001f;
	// mnist_model.train_pipelined(train_images, train_labels, 5, 32);

	auto tok = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> ms_double = tok - tik;
	std::cout << ms_double.count() << "ms \n";

	mnist_model.test(test_images, test_labels, 32);

	return 0;
}
